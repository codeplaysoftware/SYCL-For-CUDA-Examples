/***************************************************************************
 *
 *  Copyright (C) Codeplay Software Ltd.
 *
 *  Licensed under the Apache License, Version 2.0 (the "License");
 *  you may not use this file except in compliance with the License.
 *  You may obtain a copy of the License at
 *
 *      http://www.apache.org/licenses/LICENSE-2.0
 *
 *  Unless required by applicable law or agreed to in writing, software
 *  distributed under the License is distributed on an "AS IS" BASIS,
 *  WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 *  See the License for the specific language governing permissions and
 *  limitations under the License.
 *
 *  Codeplay's SYCL-For-CUDA-Examples
 *
 *  sgemm.cu
 *
 *  Description:
 *    SGEMM operation in CUDA
 **************************************************************************/
#include <algorithm>
#include <iostream>
#include <vector>

#include <hipblas.h>
#include <hip/hip_runtime.h>

#define CHECK_ERROR(FUNC) checkCudaErrorMsg(FUNC, " " #FUNC)

void inline checkCudaErrorMsg(hipblasStatus_t status, const char *msg) {
  if (status != HIPBLAS_STATUS_SUCCESS) {
    std::cout << msg << " - " << status << std::endl;
    exit(EXIT_FAILURE);
  }
}

void inline checkCudaErrorMsg(hipError_t status, const char *msg) {
  if (status != hipSuccess) {
    std::cout << msg << " - " << status << std::endl;
    exit(EXIT_FAILURE);
  }
}

int main() {
  constexpr size_t WIDTH = 1024;
  constexpr size_t HEIGHT = 1024;
  constexpr float ALPHA = 1.0f;
  constexpr float BETA = 0.0f;

  std::vector<float> h_A(WIDTH * HEIGHT), h_B(WIDTH * HEIGHT),
      h_C(WIDTH * HEIGHT);

  std::cout << "Size: " << h_C.size() << std::endl;
  float *d_A, *d_B, *d_C;

  // A is an identity matrix
  std::fill(std::begin(h_A), std::end(h_A), 0.0f);
  for (size_t i = 0; i < WIDTH; i++) {
    h_A[i * WIDTH + i] = 1.0f;
  }

  // B is a matrix fill with 1
  std::fill(std::begin(h_B), std::end(h_B), 1.0f);

  const size_t numBytes = WIDTH * HEIGHT * sizeof(float);

  CHECK_ERROR(hipMalloc((void **)&d_A, numBytes));
  CHECK_ERROR(hipMalloc((void **)&d_B, numBytes));
  CHECK_ERROR(hipMalloc((void **)&d_C, numBytes));

  CHECK_ERROR(hipMemcpy(d_A, h_A.data(), numBytes, hipMemcpyHostToDevice));
  CHECK_ERROR(hipMemcpy(d_B, h_B.data(), numBytes, hipMemcpyHostToDevice));

  hipblasHandle_t handle;
  CHECK_ERROR(hipblasCreate(&handle));

  // C = A * B
  CHECK_ERROR(hipblasSgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, WIDTH, HEIGHT,
                          WIDTH, &ALPHA, d_A, WIDTH, d_B, WIDTH, &BETA, d_C,
                          WIDTH));

  CHECK_ERROR(hipMemcpy(h_C.data(), d_C, numBytes, hipMemcpyDeviceToHost));

  // C must be all ones
  const bool allEqual = std::all_of(std::begin(h_C), std::end(h_C),
                                    [](float num) { return num == 1; });

  if (!allEqual) {
    std::cout << " Incorrect result " << std::endl;
  } else {
    std::cout << " Correct! " << std::endl;
  }

  CHECK_ERROR(hipblasDestroy(handle));
  CHECK_ERROR(hipFree(d_A));
  CHECK_ERROR(hipFree(d_B));
  CHECK_ERROR(hipFree(d_C));

  return allEqual ? EXIT_SUCCESS : EXIT_FAILURE;
}
