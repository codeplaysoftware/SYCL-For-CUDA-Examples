/***************************************************************************
 *
 *  Copyright (C) Codeplay Software Ltd.
 *
 *  Licensed under the Apache License, Version 2.0 (the "License");
 *  you may not use this file except in compliance with the License.
 *  You may obtain a copy of the License at
 *
 *      http://www.apache.org/licenses/LICENSE-2.0
 *
 *  Unless required by applicable law or agreed to in writing, software
 *  distributed under the License is distributed on an "AS IS" BASIS,
 *  WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 *  See the License for the specific language governing permissions and
 *  limitations under the License.
 *
 *  Codeplay's SYCL-For-CUDA-Examples
 *
 *  vadd_cuda.cu
 *
 *  Description:
 *    Vector addition in CUDA
 **************************************************************************/

#include <hip/hip_runtime.h>
#include <array>

// CUDA kernel. Each thread takes care of one element of c
template<class T>
__global__ void vecAdd(T *a, T *b, T *c, int n)
{
    // Get our global thread ID
    int id = blockIdx.x*blockDim.x+threadIdx.x;
 
    // Make sure we do not go out of bounds
    if (id < n)
        c[id] = a[id] + b[id];
}
 
template <typename T, size_t N>
void simple_vadd_cuda(const std::array<T, N>& VA, const std::array<T, N>& VB,
                 std::array<T, N>& VC) {
    // Device input vectors
    T *d_a;
    T *d_b;
    //Device output vector
    T *d_c;
 
    // Size, in bytes, of each vector
    const size_t bytes = N*sizeof(T);
 
    // Allocate memory for each vector on GPU
    hipMalloc(&d_a, bytes);
    hipMalloc(&d_b, bytes);
    hipMalloc(&d_c, bytes);
 
    // Copy host vectors to device
    hipMemcpy( d_a, VA.data(), bytes, hipMemcpyHostToDevice);
    hipMemcpy( d_b, VB.data(), bytes, hipMemcpyHostToDevice);
 
    int blockSize, gridSize;
 
    // Number of threads in each thread block
    blockSize = 1024;
 
    // Number of thread blocks in grid
    gridSize = (int)ceil((float)N/blockSize);
 
    // Execute the kernel
    vecAdd<<<gridSize, blockSize>>>(d_a, d_b, d_c, N);
 
    // Copy array back to host
    hipMemcpy( VC.data(), d_c, bytes, hipMemcpyDeviceToHost );
 
    // Release device memory
    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_c);

}


template void simple_vadd_cuda<float, 4>(const std::array<float, 4>& VA, const std::array<float, 4>& VB,
                 std::array<float, 4>& VC);
template void simple_vadd_cuda<int, 4>(const std::array<int, 4>& VA, const std::array<int, 4>& VB,
                 std::array<int, 4>& VC);

